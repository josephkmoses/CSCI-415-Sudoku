#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/types.h>
#include <cstdio>
#include <math.h>
#include <sys/time.h>
#include <iostream>
#include <iomanip>
#include <string>
#include <algorithm>

using namespace std;

__device__ bool valueAllowedCheck(int row, int col, int value, int* puzzle)
{
		int i; //loop vairable

		for(i = 0; i < 9; i++)
		{
				if(puzzle[row * 9 + i] == value) //check cells in the row
				{
						return false;
				}
				else if(puzzle[col + i * 9] == value) //check cells in the column
				{
						return false;
				}
				else if(puzzle[(row/3*3+i%3) * 9 + (col/3*3+i/3) ] == value) //check the subsquare
				{
						return false;
				}
		}

		return true; //the value works
}

__device__ bool solve(int row, int col, int* puzzle, int counter, int startValue, bool* finished)
{		
	if(counter == 81) //every cell has been visted
	{
			return true;
	}
	if(*finished)
	{
			return true;
	}

	//loop of column and rows
	if(++col == 9)
	{
			col = 0;
			if(++row == 9)
			{
					row = 0;
			}
	}

	//skip cells that are already filled
	if(puzzle[row * 9 + col] != 0)
	{
			return solve(row, col, puzzle, counter+1, startValue, finished);
	}

	//if the cell is empty
	for(int val = 1; val <= 9; val++)
	{
			//loop through the values being checked recursively
			if(++startValue == 10)
			{
					startValue = 1;
			}

			//check if the value is allowed
			if(valueAllowedCheck(row, col, startValue, puzzle))
			{
					puzzle[row * 9 + col] = startValue; //record if it is

					if(solve(row, col, puzzle, counter+1, startValue, finished)) //solve the next cell
					{
							return true;
					}
			}
	}
	puzzle[row * 9 + col] = 0; //set to zero if backtracking
	return false;
}


	
__global__ void parallelSudoku(int* puzzle, bool* finished, int* result)
{
	int i = threadIdx.x;	//the thread id
	int j = threadIdx.y;
    int startVal = (blockIdx.x * blockDim.x + threadIdx.x) % 9 +1; //Starting value (1-9) N
	
	__shared__ bool* sharedFinish = *finished;
	
	bool finishedTemp;
	int resultTemp;
	
	int puzzleArray [81];
	
	for(int i =0; i < 81; i++)
	{
		puzzleArray[i] = puzzle[i];
	}

	if(solve(i,j,puzzleArray,0,startVal, sharedFinish)) 
	{
		if(!*sharedFinish)//none of the threads have finished the puzzle
		{
			finishedTemp = true;
			finished = &finishedTemp;
			
			for(int i =0; i < 81; i++)
			{
				puzzle[i] = puzzleArray[i];
			}
			
			resultTemp = 1;
			(*result) = resultTemp;
		}
	}
	else
	{
		if(!*sharedFinish)//none of the threads have finished the puzzle
		{
			finishedTemp = false;
			finished = &finishedTemp;
		}
	}

}

void printPuzzle (int* puzzle ) {
    int i;
    int count =0;
    int mod;

    for(i =0; i< 81; i++) {
      mod = count % 3;
      if(count > 8){
        count =0;
        cout << "\n";
      }
      else if (mod == 0)
      {
        cout << "  ";
      }
        count ++;
        cout << puzzle[i];
        cout << " ";
    }

	cout << "\n";
}

// Copied from assignment 1
// BEGIN: timing and error checking routines (do not modify)

// Returns the current time in microseconds
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
        std::cout << std::setprecision(5);	
	std::cout << name << ": " << ((float) (end_time - start_time)) / (1000 * 1000) << " sec\n";
	return end_time - start_time;
}

void checkErrors(const char label[])
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

// END: timing and error checking routines (do not modify)


int main() {
      //clear easy puzzle
          int* easyPuzzle = (int*)malloc(81*sizeof(int));
          int inputEasy[81] = {0,7,0,  0,5,0,  8,0,6,  //row 1
                              8,0,0,  1,7,0,  9,5,2,  //row 2
                              0,2,5,  0,4,6,  3,0,0,    //row 3

                              0,0,0,  0,0,2,  0,0,0,   //row 4
                              0,0,4,  0,9,0,  2,0,0,   //row 5
                              0,0,0,  3,0,0,  0,0,0,    //row 6

                              0,0,8,  5,3,0,  6,1,0,    //row 7
                              3,5,1,  0,8,4,  0,0,9,    //row 8
                              4,0,7,  0,2,0,  0,0,0   //row 9
                           };

          //meduim
          int* meduimPuzzle = (int*)malloc(81*sizeof(int));
          int inputMeduim[81] = { 0,7,6,  3,0,0,  0,0,4,  //row 1
                              0,0,0,  0,1,0,  3,9,0,  //row2
                              3,2,9,  8,0,0,  5,0,0,   //row3

                              0,0,0,  0,0,2,  0,8,0,    //row4
                              2,0,0,  0,0,0,  0,0,1,  //row5
                              0,8,0,  9,0,0,  0,0,0,   //row6

                              0,0,2,  0,0,8,  6,4,9,  //row7
                              0,6,7,  0,4,0,  0,0,0,  //row8
                              9,0,0,  0,0,6,  7,5,0  //row9
                              };
        int* hardPuzzle = (int*)malloc(81*sizeof(int));
        int inputHard[81] = {0,0,3,  0,9,0,   0,5,0,  //row1
                          0,0,9,  0,0,2,  0,0,0,  //row2
                          0,0,0,  5,0,1,  8,0,0,  //row3

                          4,5,0,  0,7,3,  0,0,2,  //row4
                          0,0,0,  4,0,6,  0,0,0,  //row5
                          2,0,0,  1,8,0,  0,4,6,  //row6

                          0,0,2,  7,0,4,  0,0,0,  //row 7
                          0,0,0,  2,0,0,  5,0,0,  //row8
                          0,9,0,  0,1,0,  2,0,0   //row9
                          };

        int* evilPuzzle = (int*)malloc(81*sizeof(int));
        int inputEvil[81]= {6,0,0, 0,0,8,  9,4,0, //row1
                          9,0,0,  0,0,6,  1,0,0, //row2
                          0,7,0,  0,4,0,  0,0,0,  ///row3

                          2,0,0,  6,1,0,  0,0,0, //row4
                          0,0,0,  0,0,0,  2,0,0,  //row5
                          0,8,9,  0,0,2,  0,0,0,   //row6

                          0,0,0,  0,6,0,  0,0,5,  //row7
                          0,0,0,  0,0,0,  0,3,0,  //row8
                          8,0,0,  0,0,1,  6,0,0 //row9
                      };
      int* unsolveable = (int*)malloc(81*sizeof(int));
      int inputUnsolve[81] =   {1,2,3,   4,5,6,  7,8,0,  //row1
                                0,0,0,  0,0,0,  0,0,2,   //row2
                                0,0,0,  0,0,0,  0,0,3,  //row3

                                0,0,0,  0,0,0,  0,0,4,   //row4
                                0,0,0,  0,0,0,  0,0,5,   //row5
                                0,0,0,  0,0,0,  0,0,6,    //row6

                                0,0,0,  0,0,0,  0,0,7,  //row7
                                0,0,0,  0,0,0,  0,0,8,  //row8
                                0,0,0,  0,0,0,  0,0,9   //row9
                                };


  int i;
  for(i =0; i < 81; i++) {
    easyPuzzle[i] = inputEasy[i];
    meduimPuzzle[i] = inputMeduim[i];
    hardPuzzle[i] = inputHard[i];
    evilPuzzle[i] = inputEvil[i];
    unsolveable[i] = inputUnsolve[i];
    }

	
	
	//host variables
	bool* h_finished = (bool*)malloc(sizeof(bool));
	bool finishedTemp = false;
	h_finished = &finishedTemp;
	int* h_puzzle = (int*)malloc(81*sizeof(int));
	int* h_result = (int*)malloc(sizeof(int));
	int resultTemp = 0;
	h_result = &resultTemp;
	
	int* d_puzzle;
	bool* d_finished;
	int* d_result;

	long long Total_GPU_start_time = start_timer();
	
	hipMalloc((void**) &d_puzzle, 81*sizeof(int));
	checkErrors("cudaMalloc1");
	hipMalloc((void**) &d_finished, sizeof(bool));
	checkErrors("cudaMalloc2");
	hipMalloc((void**) &d_result, sizeof(int));
	checkErrors("cudaMalloc3");
	
	hipMemcpy(d_puzzle, easyPuzzle, 81*sizeof(int), hipMemcpyHostToDevice);
	checkErrors("cudaMemcpy1");
	hipMemcpy(d_finished, h_finished, sizeof(bool), hipMemcpyHostToDevice);
	checkErrors("cudaMemcpy2");
	hipMemcpy(d_result, h_result, sizeof(int), hipMemcpyHostToDevice);
	checkErrors("cudaMemcpy3");
	
	size_t stackSize = 30000;
	hipDeviceSetLimit(hipLimitStackSize, stackSize);
	
	checkErrors("stack size");
	
	dim3 threadsPerBlock(9,9);
	
	parallelSudoku<<<1 , threadsPerBlock>>>(d_puzzle, d_finished, d_result);
	checkErrors("kernel error");
	hipDeviceSynchronize();
	
	hipMemcpy(h_finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost);
	checkErrors("cudaMemcpy4");
	hipMemcpy(h_puzzle, d_puzzle, 81*sizeof(int), hipMemcpyDeviceToHost);
	checkErrors("cudaMemcpy5");
	hipMemcpy(h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
	checkErrors("cudaMemcpy6");
	
	long long GPU_total_run_time = stop_timer(Total_GPU_start_time, "\nGPU Total Run Time");
	
	if(*h_result == 1)
	{
		cout << "Solved\n";
	}
	else
	{
		cout << "Unsolved\n";
	}
	
	printPuzzle(h_puzzle); 
	
	return 0;
	
}
